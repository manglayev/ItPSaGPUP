#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>

#define THREADS 5
#define BLOCKS 1

template <typename T> class Array
{
  private:
	  T* ptr;
	  int size;

  public:
    __host__ __device__ Array(T arr[], int s);
    __host__ __device__ void print();
};

template <class T> __host__ __device__ Array<T>::Array(T arr[], int s)
{
	ptr = new T[s];
	size = s;
	for (int i = 0; i < size; i++)
		ptr[i] = arr[i];
}

template <class T> __host__ __device__ void Array<T>::print()
{
    for (int i = 0; i < size; i++)
      printf(" %d", *(ptr + i));
    printf("\n");
}

__global__ void globalFunction(int *x)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < 1)
  {
    int arr[THREADS] = { 1, 2, 3, 4, 5 };
    Array<int> array(arr, THREADS);
    printf("PRINT FROM KERNEL:\n");
    array.print();
  }
}

int main()
{
  //C++ version
  int arr[THREADS] = { 1, 2, 3, 4, 5 };
  Array<int> array(arr, THREADS);
  printf("PRINT FROM HOST:\n");
  array.print();
  //CUDA version
  int *dev_a;
  hipMallocManaged(&dev_a, sizeof(int));
  dev_a[0] = THREADS;
  globalFunction<<<BLOCKS, THREADS>>>(dev_a);
  hipDeviceSynchronize();
  printf("AFTER KERNEL %d = %d\n", THREADS, dev_a[0]);
  hipFree(dev_a);
  return 0;
}